#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdint>
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>

#include "CudaInputArray.h"

/// <summary>
/// Number of possible input chars. 0-9 and A-Z
/// </summary>
constexpr uint32_t possibleCharacterCount = 36;

#include <vector>
#include <string>

using usedCharacterCount_t = int8_t;
using usedCharacterMask_t = uint32_t;

struct AnalyzedWord
{
    /// <summary>
    /// The actually used characters as bitmask
    /// </summary>
    usedCharacterMask_t usedMask;

    /// <summary>
    /// The mask for all characters that have at least one character left
    /// </summary>
    usedCharacterMask_t remainingMask;

    /// <summary>
    /// The usages per character
    /// </summary>
    usedCharacterCount_t counts[possibleCharacterCount];

    /// <summary>
    /// The id of the word
    /// </summary>
    int32_t wordId;

    /// <summary>
    /// the length of the word
    /// </summary>
    size_t length;

    /// <summary>
    /// The remaining count per character to fullfill the anagram
    /// </summary>
    usedCharacterCount_t remaining[possibleCharacterCount];

    void initAnagram(const std::string& text)
    {
        memset(&counts, 0, sizeof(counts));
        usedMask = 0;
        length = text.size();

        for (auto c : text)
        {
            if (isalnum(c))
            {
                auto ci = toIndex(c);
                counts[ci]++;

                auto cu = std::min(ci, (char)31);
                usedMask |= 1 << cu;
            }
        }

        memcpy(&remaining, counts, sizeof(counts));
        remainingMask = usedMask;
    }

    bool initWord(const AnalyzedWord& anagram, const std::string& text)
    {
        memset(&counts, 0, sizeof(counts));
        memcpy(&remaining, anagram.remaining, sizeof(remaining));
        remainingMask = anagram.usedMask;
        usedMask = 0;

        for (auto c : text)
        {
            auto ci = toIndex(c);
            counts[ci]++;

            if (remaining[ci] == 0)
            {
                return false;
            }
            
            usedMask |= 1 << (std::min(ci, (char)31));
            if (remaining[ci] > 1)
            {
                remaining[ci]--;
            }
            else
            {
                remaining[ci] = 0;
                if (ci < 31)
                {
                    remainingMask ^= 1 << ci;
                }
            }
        }

        return true;
    }

private:
    inline char toIndex(char c)
    {
        if (c >= 'A')
        {
            return toupper(c) - 'A' + 10;
        }

        return c - '0';
    }
};

struct partialAnagramEntry
{
    partialAnagramEntry()
    {
    }

    /// <summary>
    /// Constructor from a singleWord
    /// </summary>
    /// <param name="word"></param>
    partialAnagramEntry(const AnalyzedWord& word)
    {
        initFromAnalyzedWord(word);
    }

    /// <summary>
    /// bitflags for all characters that have an expected count of 0
    /// </summary>
    usedCharacterMask_t doNotUseMask;

    /// <summary>
    /// Index of the previous entry for combinations. -1 for single words
    /// </summary>
    int32_t previousEntry;

    /// <summary>
    /// The id of the word in a global array
    /// </summary>
    int32_t wordId;

    /// <summary>
    /// the counts per character
    /// </summary>
    usedCharacterCount_t counts[possibleCharacterCount];

    /// <summary>
    /// copy data from an analysed word
    /// </summary>
    /// <param name="word">the word</param>
    void initFromAnalyzedWord(const AnalyzedWord& word)
    {
        memcpy(&counts, word.remaining, sizeof(counts));
        doNotUseMask = ~word.remainingMask;
        wordId = word.wordId;
        previousEntry = -1;
    }

    /// <summary>
    /// join existing data with a new word
    /// </summary>
    __host__ __device__
    void joinWord(const partialAnagramEntry& entry, const AnalyzedWord& word, int index)
    {
        previousEntry = index;
        doNotUseMask = entry.doNotUseMask;
        wordId = word.wordId;
        for (int i = 0; i < possibleCharacterCount; i++)
        {
            auto count = entry.counts[i] - word.counts[i];
            counts[i] = count;
            if (count == 0)
            {
                doNotUseMask |= i < 32 ? (1 << i) : (1 << 31);
            }
        }
    }
};

constexpr auto outputSizeBuffer = 1<<18;
struct Soutput
{
    int results[outputSizeBuffer];
    partialAnagramEntry output[outputSizeBuffer];
    int outputCount;
    int overflow;
    int resultCount;
    int callCount;
};

__device__ __host__ void doHandleBlock(const AnalyzedWord* word, int index, const partialAnagramEntry & entry, Soutput* output)
{
#ifdef  __CUDA_ARCH__
    atomicAdd(&output->callCount, 1);
#else
    output->callCount++;
#endif

    if (entry.doNotUseMask & word->usedMask)
    {
        return;
    }

    usedCharacterCount_t maskOfSum = 0;
    for (size_t i = 0; i < possibleCharacterCount; i++)
    {
        maskOfSum |= (entry.counts[i] - word->counts[i]);
    }

    if (maskOfSum == 0)
    {
#ifdef  __CUDA_ARCH__
        auto resultIndex = atomicAdd(&output->resultCount, 1);
#else
        auto resultIndex = output->resultCount++;
#endif
        if (resultIndex >= outputSizeBuffer)
        {
            output->overflow = true;
            return;
        }

        // valid result
        output->results[resultIndex] = index;
    }
    else if (maskOfSum > 0)
    {
        // valid combination
#ifdef  __CUDA_ARCH__
        auto& target = output->output[atomicAdd(&output->outputCount, 1)];
#else
        auto& target = output->output[output->outputCount++];
#endif
        if (output->outputCount >= outputSizeBuffer)
        {
            output->overflow = true;
            return;
        }

        target.joinWord(entry, *word, index);
    }
}

__global__ void handleBlock(AnalyzedWord* word, const partialAnagramEntry* block, Soutput * output)
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    const partialAnagramEntry& entry = block[index];
    doHandleBlock(word, index, entry, output);
}

using AnagramBlock = CudaInputArray<partialAnagramEntry>;

CudaInputArray<partialAnagramEntry> parts;
std::vector<std::string> strings;

void reportResult(int wordId, int moreResults)
{
    std::vector<int> wordIds{ wordId };
    while (moreResults >= 0)
    {
        wordIds.push_back(parts.data[moreResults].wordId);
        moreResults = parts.data[moreResults].previousEntry;
    }

    for (auto it = wordIds.rbegin(); it != wordIds.rend(); ++it)
    {
        std::cout << strings[*it] << " ";
    }

    std::cout << std::endl;
}

Soutput output;
hipError_t cudaAnagram(AnalyzedWord* word, AnagramBlock & block, Soutput* output);
int main()
{
    AnalyzedWord anagram;
    AnalyzedWord current;
    int totalResults = 0;

    std::ifstream infile("C:\\Users\\Ruttmann\\source\\repos\\AnagramCuda\\wordlist.txt");
    // anagram.initAnagram("Best Secret Aschheim");
    anagram.initAnagram("Best Secret Aschheim");
    // anagram.initAnagram("012");

    std::string s;
    output.callCount = 0;
    while (std::getline(infile, s))
    {
        if (s.size() <= 2)
        {
            continue;
        }

        if (current.initWord(anagram, s))
        {
//            std::cerr << "consider " << s << std::endl;
            current.wordId = (decltype(current.wordId))strings.size();
            strings.emplace_back(std::move(s));

            if (current.remainingMask == 0)
            {
                reportResult(current.wordId, -1);
                continue;
            }

            output.resultCount = 0;
            output.outputCount = 0;
            output.overflow = 0;
#if true
            auto cudaStatus = cudaAnagram(&current, parts, &output);
            if (cudaStatus != hipSuccess)
            {
                std::cerr << "cudaAnagram failed " << cudaStatus << std::endl;
                return 1;
            }
#else
            for (size_t i = 0; i < parts.data.size(); i++)
            {
                doHandleBlock(&current, i, parts.data[i], &output);
            }
#endif

            if (output.overflow)
            {
                reportResult(current.wordId, -1);
                std::cerr << "Too many results: " << output.outputCount << " " << output.resultCount << std::endl;
            }

            parts.data.emplace_back(current);

            for (size_t i = 0; i < std::min(outputSizeBuffer, output.outputCount); i++)
            {
                parts.data.push_back(output.output[i]);
            }

            for (size_t i = 0; i < ::min(outputSizeBuffer, output.resultCount); i++)
            {
                totalResults++;
//                reportResult(current.wordId, output.results[i]);
            }
        }
    }

    std::cout << "found " << totalResults << std::endl;
}

template<typename T>
void uploadToCuda(hipError_t& error, T& devicePtr, const T data)
{
    if (error != hipSuccess)
    {
        return;
    }

    constexpr auto size = sizeof(decltype(*devicePtr));
    error = hipMalloc((void**)&devicePtr, size);
    if (error != hipSuccess)
    {
        return;
    }

    error = hipMemcpy(devicePtr, data, size, hipMemcpyHostToDevice);
}

hipError_t cudaAnagram(AnalyzedWord* word, AnagramBlock& block, Soutput* output)
{
    AnalyzedWord* dev_word = NULL;
    Soutput* dev_output = NULL;

    // Choose which GPU to run on, change this on a multi-GPU system.
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << std::endl;
        return cudaStatus;
    }

    uploadToCuda(cudaStatus, dev_word, word);
    uploadToCuda(cudaStatus, dev_output, output);
    block.Upload();

    auto blockCount = block.data.size() / 1024 + 1;
    if (blockCount > 1)
    {
        handleBlock <<<blockCount - 1, 1024>>> (dev_word, block.dev_memory, dev_output);
    }

    handleBlock<<<1, block.data.size() % 1024>>>(dev_word, block.dev_memory, dev_output);

    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(output, dev_output, sizeof(Soutput), hipMemcpyDeviceToHost);

    hipFree(dev_word);
    hipFree(dev_output);

    return cudaStatus;
}
