
#include <hip/hip_runtime.h>
﻿
#if false

#include "cuda_runtime.h"
#include "device_launch_parameters.h"

#include <stdio.h>
#include <cstdint>
#include <algorithm>
#include <vector>
#include <iostream>
#include <fstream>

#include <execution>

#include "CudaInputArray.h"

/// <summary>
/// Number of possible input chars. 0-9 and A-Z
/// </summary>
constexpr uint32_t possibleCharacterCount = 36;

/// <summary>
/// Number of threads inside cuda hardware
/// </summary>
constexpr auto cudaThreadCount = 512;

#include <vector>
#include <string>
#include <mutex>

using usedCharacterCount_t = int8_t;
using usedCharacterMask_t = uint32_t;

struct AnalyzedWord
{
    /// <summary>
    /// The actually used characters as bitmask
    /// </summary>
    usedCharacterMask_t usedMask;

    /// <summary>
    /// The mask for all characters that have at least one character left
    /// </summary>
    usedCharacterMask_t remainingMask;

    /// <summary>
    /// The usages per character
    /// </summary>
    usedCharacterCount_t counts[possibleCharacterCount];

    /// <summary>
    /// The id of the word
    /// </summary>
    int32_t wordId;

    /// <summary>
    /// the length of the word
    /// </summary>
    size_t length;

    /// <summary>
    /// the length until the anagram is fullfilled
    /// </summary>
    size_t restLength;

    /// <summary>
    /// The remaining count per character to fullfill the anagram
    /// </summary>
    usedCharacterCount_t remaining[possibleCharacterCount];

    /// <summary>
    /// Check these character indices (the rest is covered by the mask)
    /// </summary>
    int checkCharacters[possibleCharacterCount];

    int checkCharacterCount;

    void initAnagram(const std::string& text)
    {
        memset(&counts, 0, sizeof(counts));
        usedMask = 0;
        length = text.size();
        restLength = length;
        checkCharacterCount = 0;

        for (auto c : text)
        {
            if (isalnum(c))
            {
                auto ci = toIndex(c);
                if (counts[ci] == 0)
                {
                    checkCharacters[checkCharacterCount++] = ci;
                }
                
                counts[ci]++;

                auto cu = std::min(ci, (char)31);
                usedMask |= 1 << cu;
            }
        }

        memcpy(&remaining, counts, sizeof(counts));
        remainingMask = usedMask;
    }

    bool initWord(const AnalyzedWord& anagram, const std::string& text)
    {
        memset(&counts, 0, sizeof(counts));
        memcpy(&remaining, anagram.remaining, sizeof(remaining));
        remainingMask = anagram.usedMask;
        usedMask = 0;
        checkCharacterCount = anagram.checkCharacterCount;
        memcpy(checkCharacters, anagram.checkCharacters, sizeof(checkCharacters));
        length = text.size();
        restLength = anagram.length - length;

        for (auto c : text)
        {
            auto ci = toIndex(c);
            counts[ci]++;

            if (remaining[ci] == 0)
            {
                return false;
            }
            
            usedMask |= 1 << (std::min(ci, (char)31));
            if (remaining[ci] > 1)
            {
                remaining[ci]--;
            }
            else
            {
                remaining[ci] = 0;
                if (ci < 31)
                {
                    remainingMask ^= 1 << ci;
                }
            }
        }

        return true;
    }

private:
    inline char toIndex(char c)
    {
        if (c >= 'A')
        {
            return toupper(c) - 'A' + 10;
        }

        return c - '0';
    }
};

struct partialAnagramEntry
{
    partialAnagramEntry()
    {
    }

    /// <summary>
    /// Constructor from a singleWord
    /// </summary>
    /// <param name="word"></param>
    partialAnagramEntry(const AnalyzedWord& word)
    {
        initFromAnalyzedWord(word);
    }

    /// <summary>
    /// bitflags for all characters that have an expected count of 0
    /// </summary>
    usedCharacterMask_t doNotUseMask;

    /// <summary>
    /// Index of the previous entry for combinations. -1 for single words
    /// </summary>
    int32_t previousEntry;

    /// <summary>
    /// The id of the word in a global array
    /// </summary>
    int32_t wordId;

    /// <summary>
    /// the counts per character
    /// </summary>
    usedCharacterCount_t counts[possibleCharacterCount];

    /// <summary>
    /// total sum of characters
    /// </summary>
    int32_t restLength;

    /// <summary>
    /// copy data from an analysed word
    /// </summary>
    /// <param name="word">the word</param>
    void initFromAnalyzedWord(const AnalyzedWord& word)
    {
        memcpy(&counts, word.remaining, sizeof(counts));
        doNotUseMask = ~word.remainingMask;
        wordId = word.wordId;
        previousEntry = -1;
    }

    /// <summary>
    /// join existing data with a new word
    /// </summary>
    __host__ __device__
    void joinWord(const partialAnagramEntry& entry, const AnalyzedWord& word, int index)
    {
        previousEntry = index;
        doNotUseMask = entry.doNotUseMask;
        wordId = word.wordId;
        for (int i = 0; i < possibleCharacterCount; i++)
        {
            auto count = entry.counts[i] - word.counts[i];
            counts[i] = count;
            if (count == 0)
            {
                doNotUseMask |= i < 32 ? (1 << i) : (1 << 31);
            }
        }
    }
};

constexpr auto outputSizeBuffer = 512;
struct Soutput
{
    int outputCount;
    int overflow;
    int resultCount;
    int callCount;
    int results[outputSizeBuffer];
    partialAnagramEntry output[outputSizeBuffer + cudaThreadCount];
};

__device__ __host__ void doHandleBlock(const AnalyzedWord word, int index, const partialAnagramEntry & entry, Soutput* output)
{
#ifdef  __CUDA_ARCH__
    atomicAdd(&output->callCount, 1);
#else
    output->callCount++;
#endif

    if (entry.doNotUseMask & word.usedMask)
    {
        return;
    }

#if false
    usedCharacterCount_t maskOfSum = 0;
    for (auto i = 0; i < word.checkCharacterCount; i++)
    {
        auto ci = word.checkCharacters[i];
        maskOfSum |= (entry.counts[ci] - word.counts[ci]);
    }
#else
    usedCharacterCount_t maskOfSum = 0;
    for (size_t i = 0; i < possibleCharacterCount; i++)
    {
        maskOfSum |= (entry.counts[i] - word.counts[i]);
    }
#endif

    if (maskOfSum == 0)
    {
#ifdef  __CUDA_ARCH__
        auto resultIndex = atomicAdd(&output->resultCount, 1);
#else
        auto resultIndex = output->resultCount++;
#endif
        if (resultIndex >= outputSizeBuffer)
        {
            output->overflow = true;
            return;
        }

        // valid result
        output->results[resultIndex] = index;
    }
    else if ((maskOfSum > 0) && (entry.restLength - word.length > 2))
    {
        // valid combination
#ifdef  __CUDA_ARCH__
        auto& target = output->output[atomicAdd(&output->outputCount, 1)];
#else
        auto& target = output->output[output->outputCount++];
#endif
        if (output->outputCount >= outputSizeBuffer + cudaThreadCount)
        {
            output->overflow = true;
            return;
        }

        target.joinWord(entry, word, index);
    }
}

__global__ void handleBlock(const AnalyzedWord* word, int max, const partialAnagramEntry* block, Soutput * output)
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= max)
    {
        return;
    }

    const partialAnagramEntry& entry = block[index];
    doHandleBlock(*word, index, entry, output);
}

using AnagramBlock = CudaInputArray<partialAnagramEntry>;

CudaInputArray<partialAnagramEntry> parts;
std::vector<std::string> strings;

void reportResult(int wordId, int moreResults)
{
    std::vector<int> wordIds{ wordId };
    while (moreResults >= 0)
    {
        wordIds.push_back(parts.data[moreResults].wordId);
        moreResults = parts.data[moreResults].previousEntry;
    }

    for (auto it = wordIds.rbegin(); it != wordIds.rend(); ++it)
    {
        std::cout << strings[*it] << " ";
    }

    std::cout << std::endl;
}

Soutput g_output;
int totalResults;
long long totalCalls = 0;
cudaError_t cudaAnagram(AnalyzedWord* word, AnagramBlock & block, Soutput* output);
void handleOutputBlock(const AnalyzedWord* word, AnagramBlock& block, Soutput* output);

int main()
{
    AnalyzedWord anagram;
    AnalyzedWord current;

    std::ifstream infile("C:\\Users\\Ruttmann\\source\\repos\\AnagramCuda\\wordlist.txt");
    // anagram.initAnagram("Best Secret Aschheim");
    anagram.initAnagram("Best Secret Aschheim");
    // anagram.initAnagram("012");

    std::string s;
    g_output.callCount = 0;
    totalResults = 0;
    parts.data.reserve(500000);

    while (std::getline(infile, s))
    {
        if (s.size() <= 2)
        {
            continue;
        }

        if (current.initWord(anagram, s))
        {
//            std::cerr << "consider " << s << std::endl;
            current.wordId = (decltype(current.wordId))strings.size();
            strings.emplace_back(std::move(s));

            if (current.remainingMask == 0)
            {
                reportResult(current.wordId, -1);
                continue;
            }

            g_output.resultCount = 0;
            g_output.outputCount = 0;
            g_output.overflow = 0;
#if false
            auto cudaStatus = cudaAnagram(&current, parts, &g_output);
            if (cudaStatus != cudaSuccess)
            {
                std::cerr << "cudaAnagram failed " << cudaStatus << std::endl;
                return 1;
            }
#else
            auto& a = parts.data;
            std::mutex m;
            std::for_each(
                std::execution::par_unseq,
                std::begin(a), 
                std::end(a), 
                [&m, &current](auto & value)
                {
                    {
                        std::lock_guard<std::mutex> guard(m);
                        totalCalls++;
                    }

                    int idx = 0;
                    doHandleBlock(current, idx, value, &g_output);
                });

            auto end = parts.data.size();
            for (size_t i = 0; i < end; i++)
            {
                totalCalls++;
                doHandleBlock(current, i, parts.data[i], &g_output);
                if (g_output.outputCount > outputSizeBuffer)
                {
                    handleOutputBlock(&current, parts, &g_output);
                }
            }

            handleOutputBlock(&current, parts, &g_output);
#endif
            parts.data.emplace_back(current);
        }
    }

    std::cout << "found " << totalResults << " " << parts.data.size() << " " << totalCalls << " " << g_output.callCount << std::endl;
}

template<typename T>
void uploadToCuda(cudaError_t& error, T& devicePtr, const T data)
{
    if (error != cudaSuccess)
    {
        return;
    }

    constexpr auto size = sizeof(decltype(*devicePtr));
    error = cudaMalloc((void**)&devicePtr, size);
    if (error != cudaSuccess)
    {
        return;
    }

    error = cudaMemcpy(devicePtr, data, size, cudaMemcpyHostToDevice);
}

void handleOutputBlock(const AnalyzedWord* word, AnagramBlock& block, Soutput* output)
{
    if (output->overflow)
    {
        reportResult(word->wordId, -1);
        std::cerr << "Too many results: " << output->outputCount << " " << output->resultCount << std::endl;
    }

    for (size_t i = 0; i < std::min(outputSizeBuffer + cudaThreadCount, output->outputCount); i++)
    {
        parts.data.push_back(output->output[i]);
    }

    for (size_t i = 0; i < std::min(outputSizeBuffer, output->resultCount); i++)
    {
        totalResults++;
        reportResult(word->wordId, output->results[i]);
    }

    output->resultCount = 0;
    output->outputCount = 0;
}

cudaError_t cudaAnagram(AnalyzedWord* word, AnagramBlock& block, Soutput* output)
{
    AnalyzedWord* dev_word = NULL;
    Soutput* dev_output = NULL;

    // Choose which GPU to run on, change this on a multi-GPU system.
    auto cudaStatus = cudaSetDevice(0);
    if (cudaStatus != cudaSuccess) {
        std::cerr << "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?" << std::endl;
        return cudaStatus;
    }

    constexpr auto oututExchangeSize = 16;
    uploadToCuda(cudaStatus, dev_word, word);
    uploadToCuda(cudaStatus, dev_output, output);
    block.Upload();

#if false
    auto blockSize = block.data.size();
    for (size_t offset = 0; offset < blockSize; offset += cudaThreadCount)
    {
        auto count = std::min(blockSize - offset, cudaThreadCount);
        handleBlock<<<1, count>>> (dev_word, block.dev_memory + offset, dev_output);
        cudaStatus = cudaDeviceSynchronize();

        // copy only the counters and do intermediate reporting
        cudaMemcpy(output, dev_output, oututExchangeSize, cudaMemcpyDeviceToHost);
        if (output->outputCount >= outputSizeBuffer || output->resultCount >= outputSizeBuffer)
        {
            cudaStatus = cudaMemcpy(output, dev_output, sizeof(Soutput), cudaMemcpyDeviceToHost);
            handleOutputBlock(word, block, output);
            cudaStatus = cudaMemcpy(output, dev_output, oututExchangeSize, cudaMemcpyHostToDevice);
        }
    }
#else
    handleBlock<<<block.data.size() / cudaThreadCount + 1, cudaThreadCount>>>(
        dev_word, block.data.size(), block.dev_memory, dev_output);
#endif

    cudaStatus = cudaDeviceSynchronize();
    cudaStatus = cudaMemcpy(output, dev_output, sizeof(Soutput), cudaMemcpyDeviceToHost);
    handleOutputBlock(word, block, output);

    cudaFree(dev_word);
    cudaFree(dev_output);

    return cudaStatus;
}

#endif